#include "hip/hip_runtime.h"
﻿using namespace std;
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <chrono>
#include <float.h>

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;


#define RANGE_COORDINATE_MAX 100000		// range di coordinate nel piano
#define CLUSTER_NUMBER 50			// numero di cluster da creare
#define POINT_NUMBER 5000000     // numero di punti da creare
#define CLUSTER_ATTRIBUTES 4		// numero di caratteristiche di un punto
#define POINT_ATTRIBUTES 3		// <- numero di caratteristiche di un cluster
#define ITERATION 30               // numero massimo di iterazioni
#define THREAD_FOR_BLOCK 256	// numero di thread per blocco (con gt1030 massimo 256)
#define DISTANCE 0 // parametro per la scelta della distanza -> 0: distanza euclidea -> 1: distanza di Manhattan

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/*
------------------------------------------------------------------------------
	* Metodo per un' ottima visualizzazione degli errori della GPU.
	* Credits: StackOverflow
------------------------------------------------------------------------------
*/
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
------------------------------------------------------------------------------
	* Metodo per generare un numero casuale all'interno di RANGE_COORDINATE_MAX
	* Questo metodo viene utlizzato per generare casualmente i punti.
------------------------------------------------------------------------------
*/

__host__ float random_float() {
	float x = (float)rand() * (float)rand();
	return fmod(x, RANGE_COORDINATE_MAX);
}

/*
------------------------------------------------------------------------------
	* Metodi per il calcolo della distanza fra due punti p1 e p2.
	*
	* distanza euclidea (x1,y1,x2,y2)->sqrt[(x1-x2)^2+(y1-y2)^2]
	*
	* distanza di Manhttan(x1,y1,x2,y2)->|x1-x2|+|y1-y2|
	*
	* per utilizzarli è necessario cambiare la macro DISTANCE
------------------------------------------------------------------------------
*/
__device__ float euclideanDistance(float x1, float y1, float x2, float y2) {
	return sqrt(powf((x1 - x2), 2.0) + powf((y1 - y2), 2.0));
}
__device__ float manhttanDistance(float x1, float y1, float x2, float y2) {
	return fabsf(x1 - x2) + fabsf(y1 - y2);
}
/*
– – – – – – – – – – – – – – – – – – – – – – – – – – – – – - - - - - - -– – – – – – – – – – – – – – – – – – – – – – – – – –
 * metodo per la generazione casuale della matrici dei punti e dei clusters
 *
 * ogni punto avrà la forma [c,x,y] con:
 *                                      c-> cluster di appartenenza
 *                                      x-> coordinata sull'asse x
 *                                      y-> coordinata sull'asse y
 *
 * ogni cluster avrà la forma [p,n,sx,sy] con:
 *                                          p->indice all'interno della matrice dei punti chenidentifica il centroide
 *                                          n-> numero di punti all'interno del cluster
 *                                          vx-> somma dei valori sull'asse x per il ricalco del centroide
 *                                          vy-> somma dei valori sull'asse y  per il ricalco del centroide
 *
 * i punti verranno salvati in array della forma [c0,c1,...,cn,x1,x2,...,xn,y1,y2,...,yn]
 * i cluster verrano salvati in un array della forma [p0,p1,...,pn,n1,n2,...,nn,vx1,vx2,...,vxn,vy1,vy2,...,vyn]
- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - – – – – – – – – – – – – – – – – – – – – – – – – –
*/
__host__ void generatePointCluster(float* points, float* clusters) {
	for (int j = 0; j < POINT_NUMBER; j++) {
		points[0 * POINT_NUMBER + j] = -1;
		points[1 * POINT_NUMBER + j] = random_float();
		points[2 * POINT_NUMBER + j] = random_float();
	}

	for (int j = 0; j < CLUSTER_NUMBER; j++) {
		clusters[0 * CLUSTER_NUMBER + j] = rand() % POINT_NUMBER;
		clusters[1 * CLUSTER_NUMBER + j] = 0;
		clusters[2 * CLUSTER_NUMBER + j] = 0;
		clusters[3 * CLUSTER_NUMBER + j] = 0;
	}
}
/*
– – – – – – – – – – – – – – – – – – – – – – – – – – – – – - - - - - - - – – – – - -
 * Metodo per assegnare i punti al cluster più vicino, basandosi sulla distanza
 * euclidea o la distanza di Manhattan.
 * Dato un punto da assegnare ad un cluster si va a cercare, tra tutti i cluster
 * disponibili, il cluster il quale centroide minimizza la distanza fra il punto
 * e se stesso
- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - – – – – - -
*/

__global__ void assignPointToCluster(float* points, float* clusters) {
	unsigned int point_n = threadIdx.x + blockIdx.x * blockDim.x;
	if (point_n < POINT_NUMBER) {
		float x_cluster, y_cluster = 0;
		//si recuperano le coordinate del punto da assegnare
		float x_point = points[1 * POINT_NUMBER + point_n];
		float y_point = points[2 * POINT_NUMBER + point_n];
		unsigned int best_fitting = 0;
		float old_distance = FLT_MAX;
		float actual_distance = 0;

		for (int i = 0; i < CLUSTER_NUMBER; i++) {
			//si recuperano le coordinate del centroide
			unsigned int centroid_index = clusters[0 * CLUSTER_NUMBER + i];
			x_cluster = points[1 * POINT_NUMBER + centroid_index];
			y_cluster = points[2 * POINT_NUMBER + centroid_index];

			if (DISTANCE == 0) {
				actual_distance = euclideanDistance(x_point, y_point, x_cluster, y_cluster);
			}
			else
			{
				actual_distance = manhttanDistance(x_point, y_point, x_cluster, y_cluster);
			}
			//se la distanza tra il centroide in esame è minore della vecchia distanza
		   //allora si aggiorna il migliore centroide per il punto
			if (actual_distance < old_distance) {
				best_fitting = i;
				old_distance = actual_distance;
			}
		}
		//si  crea la relazione fra il punto e il suo miglior cluster
		points[0 * POINT_NUMBER + point_n] = best_fitting;
		atomicAdd(&clusters[1 * CLUSTER_NUMBER + best_fitting], 1);
	}
}

/*
– – – – – – – – – – – – – – – – – – – – – – – – – – – – – - - - - - - - – – – – - -
 * metodo utilizzato per calcolare la somma sui due assi di ogni punti all'interno
 * di un cluster
- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - – – – – - -
*/
__global__ void calculateValue(float* points, float* clusters) {
	unsigned int point_n = threadIdx.x + blockIdx.x * blockDim.x;
	if (point_n < POINT_NUMBER) {
		unsigned int cluster_n = points[0 * POINT_NUMBER + point_n];
		atomicAdd(&clusters[2 * CLUSTER_NUMBER + cluster_n], points[1 * POINT_NUMBER + point_n]);
		atomicAdd(&clusters[3 * CLUSTER_NUMBER + cluster_n], points[2 * POINT_NUMBER + point_n]);
	}
}

/*
– – – – – – – – – – – – – – – – – – – – – – – – – – – – – - - - - - - - – – – – - -
 * metodo utilizzato per ricalcolare il centroide all'interno di tutti i cluster
- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - – – – – - -
*/
__global__ void recomputeCentroid(float* points, float* clusters) {
	unsigned int cluster_n = threadIdx.x + blockIdx.x * blockDim.x;
	float centroid_x = clusters[2 * CLUSTER_NUMBER + cluster_n] / clusters[1 * CLUSTER_NUMBER + cluster_n];
	float centroid_y = clusters[3 * CLUSTER_NUMBER + cluster_n] / clusters[1 * CLUSTER_NUMBER + cluster_n];
	unsigned int cluster_index = (unsigned int)clusters[0 * CLUSTER_NUMBER + cluster_n];
	//float x = points[1 * POINT_NUMBER + cluster_index];
	points[1 * POINT_NUMBER + cluster_index] = centroid_x;
	points[2 * POINT_NUMBER + cluster_index] = centroid_y;

}
/*
– – – – – – – – – – – – – – – – – – – – – – – – – – – – – - - - - - - - – – – – - -
 * metodo utilizzato per "eliminare" i punti dal cluster. In pratica si settano
 * i parametri sx,sy,n a 0.
- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - – – – – - -
*/
__global__ void removePoint(float* clusters) {
	unsigned int cluster_n = threadIdx.x + blockIdx.x * blockDim.x;
	clusters[1 * CLUSTER_NUMBER + cluster_n] = 0;
	clusters[2 * CLUSTER_NUMBER + cluster_n] = 0;
	clusters[3 * CLUSTER_NUMBER + cluster_n] = 0;
}


__host__ void write(float* points) {
	ofstream file;
	file.open(R"(C:\\Users\\user\\Desktop\\clusterCuda.txt)");
	for (int i = 0; i < POINT_NUMBER; i++) {
		float cluster = points[0 * POINT_NUMBER + i];
		float x = points[1 * POINT_NUMBER + i];
		float y = points[2 * POINT_NUMBER + i];
		file << x << " " << y << " " << cluster << "\n";
	}
	file.close();
}

int main()
{

	srand(time(NULL));
	cout << "----KMmeas algorithm in CUDA--" << endl;
	cout << "Random generation of points and clusters" << endl;

	float* points_host = (float*)malloc(POINT_NUMBER * POINT_ATTRIBUTES * sizeof(float));
	float* clusters_host = (float*)malloc(CLUSTER_NUMBER * CLUSTER_ATTRIBUTES * sizeof(float));
	float* points_device = 0;
	float* clusters_device = 0;
	//generazione e copia dati su device
	generatePointCluster(points_host, clusters_host);
	hipMalloc(&points_device, POINT_NUMBER * POINT_ATTRIBUTES * sizeof(float));
	hipMalloc(&clusters_device, CLUSTER_NUMBER * CLUSTER_ATTRIBUTES * sizeof(float));
	hipMemcpy(points_device, points_host, POINT_NUMBER * POINT_ATTRIBUTES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(clusters_device, clusters_host, CLUSTER_NUMBER * CLUSTER_ATTRIBUTES * sizeof(float), hipMemcpyHostToDevice);
	cout << "Press enter to start..." << endl << endl;
	cin.ignore();

	auto start_time = high_resolution_clock::now();
	for (int i = 0; i < ITERATION; i++) {
		cout << "Iteration--> " << i << endl;
		//assegnamento punti al cluster
		assignPointToCluster << < (POINT_NUMBER + THREAD_FOR_BLOCK - 1) / THREAD_FOR_BLOCK, THREAD_FOR_BLOCK >> > (points_device, clusters_device);
		gpuErrchk(hipDeviceSynchronize());
		//calcolo del valore su asse x e y utilizzato per il ricalcolo del centroide
		calculateValue << < (POINT_NUMBER + THREAD_FOR_BLOCK - 1) / THREAD_FOR_BLOCK, THREAD_FOR_BLOCK >> > (points_device, clusters_device);
		gpuErrchk(hipDeviceSynchronize());
		//ricalcolo del centroide
		recomputeCentroid << <1, CLUSTER_NUMBER >> > (points_device, clusters_device);
		gpuErrchk(hipDeviceSynchronize());
		//rimozione dei punti da ogni cluster
		removePoint << <1, CLUSTER_NUMBER >> > (clusters_device);
		gpuErrchk(hipDeviceSynchronize());
	}
	gpuErrchk(hipDeviceSynchronize());

	auto end_time = high_resolution_clock::now();
	duration<double, std::milli> total_time = end_time - start_time;
	cout << endl << "Execution time: " << total_time.count() << " ms" << endl;
	hipMemcpy(points_host, points_device, POINT_NUMBER * POINT_ATTRIBUTES * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(clusters_host, clusters_device, CLUSTER_NUMBER * CLUSTER_ATTRIBUTES * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(points_device);
	hipFree(clusters_device);

	cout << "Write points.... " << endl;
	write(points_host);
	cout << "Press enter to open the plot..." << endl;
	cin.ignore();
	system(R"(gnuplot -p -e "plot 'C:\\Users\\user\\Desktop\\clusterCuda.txt' using 1:2:3 with points palette notitle")");
	return 0;
}
